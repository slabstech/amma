
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void square(float *d_out, float *d_in){
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f * f ;
}
__global__ void cube(float *d_out, float *d_in){
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f * f * f ;
}

int main(int argc, char const *argv[]) {

  const int ARRAY_SIZE = 64;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  float h_in[ARRAY_SIZE];
  for(int i=0;i < ARRAY_SIZE;i++){
    h_in[i] = float(i);
  }

  float h_out[ARRAY_SIZE];

  float * d_in;
  float * d_out;


  cout << "Running square" << endl;
  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_out, ARRAY_BYTES);


  hipMemcpy(d_in, h_in, ARRAY_BYTES,hipMemcpyHostToDevice );

  square<<<1, ARRAY_SIZE>>>(d_out, d_in);

  hipMemcpy(h_out, d_out, ARRAY_BYTES,hipMemcpyDeviceToHost );
  for(int i=0;i < ARRAY_SIZE;i++){
    cout << h_out[i] ;
    cout << (((i%4)!=3) ? "\t" : "\n");
  }

// For cube
  cout << "\n" ;
  cout << "Running Cube" << endl;
  for(int i=0;i < ARRAY_SIZE;i++){
    h_in[i] = float(i);
  }
  hipMemcpy(d_in, h_in, ARRAY_BYTES,hipMemcpyHostToDevice );

  cube<<<1, ARRAY_SIZE>>>(d_out, d_in);
  hipMemcpy(h_out, d_out, ARRAY_BYTES,hipMemcpyDeviceToHost );
  for(int i=0;i < ARRAY_SIZE;i++){
    cout << h_out[i] ;
    cout << (((i%4)!=3) ? "\t" : "\n");
  }




  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
