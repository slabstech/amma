
#include <hip/hip_runtime.h>
#include <iostream>
//#include "StopWatch.h"
#include <sstream>
#include <string>
#include <fstream>
#include <vector>
#include <math.h>

struct Point{
	float x,y;
};

using namespace std;

int main(int argc, char **argv)
{

//	StopWatch stopWatch;
//	stopWatch.start();


	cout << "Running Task4 to find shortest path in Polygon Maps" << endl;

	if(argc !=2)
	{
		cout << "Input file not found" << endl ;
		exit(1);
	}

	string input_file_name = argv[1];

	cout << "Processing input file : " << input_file_name << endl;
	std::ifstream infile(input_file_name);
	string line;

	std::getline(infile, line);
	std::istringstream iss(line);


    Point start;
    iss >> start.x ;
    iss >> start.y ;

    std::getline(infile, line);
    std::istringstream isse(line);


    Point end;
    isse >> end.x ;
    isse >> end.y ;


    vector<vector<Point>> polygons;

    std::getline(infile, line); // empty line


    vector<Point> temp_point;
	while (std::getline(infile, line))
	{
	    std::istringstream iss(line);

	    if(line == "")
	    {
	    	polygons.push_back(temp_point) ;
	    	temp_point.clear();
	    }
	    else
	    {
	    	Point temp;
	    	iss >> temp.x ;
	    	iss >> temp.y ;

		    temp_point.push_back(temp);

	    }
	}

	cout << "total number of polygons : " << polygons.size() << endl;

	double distance = sqrt( pow((end.x - start.x),2) +  pow(( end.y - start.y),2)) ;

	cout << "Shortest possible distance(Ignoring Obstacles): " << distance << endl ;


	for(int i=0;i< polygons.size();i++){
		temp_point = polygons.at(i);
		for(int j=0;j< temp_point.size();j++){
			Point temp = temp_point.at(j);

			//cout << temp.x << " " << temp.y << endl;
		}

		//cout << "\n" ;
	}


	ofstream vis_graph_file;

	remove( "points.txt" );
	vis_graph_file.open ("points.txt",  ios::out | ios::app);


	vector<Point> vis_graph_points;


	for(int i=0;i< polygons.size();i++){
			temp_point = polygons.at(i);  // Each polygon
			for(int j=0;j< temp_point.size();j++){
				Point temp = temp_point.at(j);

				vis_graph_points.push_back(start);
				vis_graph_points.push_back(temp);

				/*

				for(int k=0 ; k < temp_point.size() ; k++ ){
					Point next_point = temp_point.at(k);
					vis_graph_points.push_back(next_point);

					//vis_graph_file << "\n" ;
				}
*/
				//vis_graph_file << "\n" ;
			}

			//vis_graph_file << "\n" ;
		}



			for(int i=0;i< polygons.size();i++){
					temp_point = polygons.at(i);  // Each polygon
					for(int j=0;j< temp_point.size();j++){
						Point temp = temp_point.at(j);

						vis_graph_points.push_back(temp);
						vis_graph_points.push_back(end);


/*
						for(int k=0 ; k < temp_point.size() ; k++ ){
							Point next_point = temp_point.at(k);
							vis_graph_points.push_back(next_point);

							//vis_graph_file << "\n" ;
						}
*/
						//vis_graph_file << "\n" ;
					}

					//vis_graph_file << "\n" ;
				}


for(int i=0; i< vis_graph_points.size(); i++){
	Point vis_points = vis_graph_points.at(i);

	 vis_graph_file << vis_points.x << " " << vis_points.y << endl;
}



	vis_graph_file.close();


//	double time = stopWatch.elapsedTime();

	//cout << "Total Execution Time : " << time << endl;

	return 0;
}
